#include "hip/hip_runtime.h"
/********************************************************************************
*          Monte-Carlo Simulation for Light Transport in 3D Volumes             *
*********************************************************************************
*                                                                               *
* Copyright (C) 2002-2008,  David Boas    (dboas <at> nmr.mgh.harvard.edu)      *
*               2008        Jay Dubb      (jdubb <at> nmr.mgh.harvard.edu)      *
*               2008        Qianqian Fang (fangq <at> nmr.mgh.harvard.edu)      *
*               2011        Ralf Gunter   (ralfgunter <at> gmail.com)           *
*                                                                               *
* License:  4-clause BSD License, see LICENSE for details                       *
*                                                                               *
********************************************************************************/

#include "main.h"
#include "logistic_rand_kernel.h"
#include "bitset2d_kernel.h"

#define MOVE(p, r, stepr) \
        (p).x = (r).x * (stepr).x; \
        (p).y = (r).y * (stepr).y; \
        (p).z = (r).z * (stepr).z

//__constant__ int4 det_loc[MAX_DETECTORS];
//__constant__ float4 media_prop[MAX_TISSUES];
__constant__ Simulation s;
__constant__ GPUMemory g;

// TODO: do away with the first argument.
__device__ void henyey_greenstein(float *t, float gg, uint8_t media_index, uint32_t photon_idx, float3 *d)
{
    float3 d0;
    float rand;
    float foo;
    float theta, stheta, ctheta;
    float phi, sphi, cphi;

    // TODO: study more closely the random functions.
    rand = rand_next_aangle(t);
    phi = 2.0 * PI * rand;
    sincosf(phi, &sphi, &cphi);

    rand = rand_next_zangle(t);

    if(gg > EPS) {
        foo = (1.0 - gg * gg) / (1.0 - gg + 2.0 * gg * rand);
        foo *= foo;
        foo = (1.0 + gg * gg - foo) / (2.0 * gg);
        theta = acosf(foo);
        stheta = sinf(theta);
        ctheta = foo;
    } else {  // If g is exactly zero, then use isotropic scattering angle
        theta = 2.0 * PI * rand;
        sincosf(theta, &stheta, &ctheta);
    }

    if(theta > 0)
        g.mom_transfer[MAD_HASH((photon_idx << 5) | media_index)] += 1 - ctheta;

    d0.x = d->x;
    d0.y = d->y;
    d0.z = d->z;
    if( d->z < 1.0 && d->z > -1.0 ) {
        d->x = stheta * (d0.x*d0.z*cphi - d0.y*sphi) * rsqrtf(1.0 - d0.z*d0.z) + d0.x * ctheta;
        d->y = stheta * (d0.y*d0.z*cphi + d0.x*sphi) * rsqrtf(1.0 - d0.z*d0.z) + d0.y * ctheta;
        d->z = -stheta * cphi * sqrtf(1.0 - d0.z*d0.z) + d0.z * ctheta;
    } else {
        d->x  = stheta * cphi;
        d->y  = stheta * sphi;
        d->z *= ctheta;
    }
}

__global__ void run_simulation(uint32_t *seed, int photons_per_thread, int iteration)
{
    __shared__ int4 det_loc[MAX_DETECTORS + MAX_TISSUES];
    float4 *media_prop = (float4 *) det_loc + MAX_DETECTORS;

    // Loop index
    int i;

    uint32_t threadIndex = LIN2D(threadIdx.x, blockIdx.x, blockDim.x);

    uint8_t media_index;   // tissue type of the current voxel
    int time;            // time elapsed since the photon was launched
    float step;
    float musr;

    // Random number generation
    float t[RAND_BUF_LEN], tnew[RAND_BUF_LEN];

    det_loc[threadIdx.x] = g.det_loc[threadIdx.x];
    det_loc[2*threadIdx.x] = g.det_loc[2*threadIdx.x];
    media_prop[threadIdx.x] = g.media_prop[threadIdx.x];
    __syncthreads();

    gpu_rng_init(t, tnew, seed, threadIndex);

    int photons_run = 0;
    while(photons_run < photons_per_thread)
    {
        uint32_t photon_idx = LIN3D(photons_run, threadIndex, iteration, photons_per_thread, (blockDim.x * gridDim.x));
        photons_run++;

        // Set the photon weight to 1 and initialize photon length parameters
        float photon_weight = 1.0;   // photon weight
        float dist = 0.0;   // distance traveled so far by the photon 
        float Lnext = s.grid.minstepsize;
        float Lresid = 0.0;

        // Direction cosines of the photon
        float3 d;
        d.x = s.src.d.x; d.y = s.src.d.y; d.z = s.src.d.z;

        // Photon position (euclidean)
        float3 r;
        r.x = s.src.r.x; r.y = s.src.r.y; r.z = s.src.r.z;

        // Photon position (grid)
        int3 p;
        MOVE(p, r, s.grid.stepr);

        // Loop until photon has exceeded its max distance allowed, or escapes
        // the grid.
        while( dist < s.max_length &&
               p.x >= 0 && p.x < s.grid.dim.x &&
               p.y >= 0 && p.y < s.grid.dim.y &&
               p.z >= 0 && p.z < s.grid.dim.z &&
               (media_index = g.media_type[LIN3D(p.x, p.y, p.z, s.grid.dim.x, s.grid.dim.y)]) != 0 )
        {
            rand_need_more(t, tnew);

            // Calculate scattering length
            Lresid = rand_next_scatlen(t);

            while( Lresid > 0.0 && dist < s.max_length &&
                   p.x >= 0 && p.x < s.grid.dim.x &&
                   p.y >= 0 && p.y < s.grid.dim.y &&
                   p.z >= 0 && p.z < s.grid.dim.z &&
                   (media_index = g.media_type[LIN3D(p.x, p.y, p.z, s.grid.dim.x, s.grid.dim.y)]) != 0 )
            {
                if(dist > Lnext && dist > s.min_length)
                {
                    time = (int) ((dist - s.min_length) * s.stepLr);

                    if( p.x >= s.grid.fbox_min.x && p.x <= s.grid.fbox_max.x &&
                        p.y >= s.grid.fbox_min.y && p.y <= s.grid.fbox_max.y &&
                        p.z >= s.grid.fbox_min.z && p.z <= s.grid.fbox_max.z &&
                        time < s.num_time_steps )
                        g.fbox[LIN(p.x, p.y, p.z, time, s.grid)] += photon_weight;

                    Lnext += s.grid.minstepsize;
                }

                musr = media_prop[media_index].x;
                step = Lresid * musr;
                // If scattering length is likely within a voxel, jump inside one voxel
                if(s.grid.minstepsize > step) {
                    Lresid = 0.0;
                } else {   // If scattering length is bigger than a voxel, then move one voxel
                    step = s.grid.minstepsize;
                    Lresid -= musr * s.grid.minstepsize;
                }

                r.x += d.x * step;
                r.y += d.y * step;
                r.z += d.z * step;
                dist += step;

                photon_weight *= expf(-(media_prop[media_index].y) * step);
                // FIXME: on 32-bits cards, this only works with up to
                //        (2^5 - 1) tissue types (indexed from 1) and
                //        2^27 photons (indexed from 0).
                g.path_length[MAD_HASH((photon_idx << 5) | media_index)] += step;

                MOVE(p, r, s.grid.stepr);
            } // Propagate photon

            // Calculate the new scattering angle using henyey-greenstein
            if(media_index != 0)
                henyey_greenstein(t, media_prop[media_index].z, media_index, photon_idx, &d);
        } // loop until end of single photon

        // Score exiting photon
        MOVE(p, r, s.grid.stepr);

        if ( p.x >= 0 && p.x < s.grid.dim.x &&
             p.y >= 0 && p.y < s.grid.dim.y &&
             p.z >= 0 && p.z < s.grid.dim.z )
        {
            media_index = g.media_type[LIN3D(p.x, p.y, p.z, s.grid.dim.x, s.grid.dim.y)];
            if(media_index == 0)
            {
                time = (int) ((dist - s.min_length) * s.stepLr);
                if( p.x >= s.grid.fbox_min.x && p.x <= s.grid.fbox_max.x &&
                    p.y >= s.grid.fbox_min.y && p.y <= s.grid.fbox_max.y &&
                    p.z >= s.grid.fbox_min.z && p.z <= s.grid.fbox_max.z &&
                    time < s.num_time_steps )
                    g.fbox[LIN(p.x, p.y, p.z, time, s.grid)] -= photon_weight;

                // Did the photon hit a detector?
                for( i = 0; i < s.det.num; i++ )
                    if( absf(p.x - det_loc[i].x) <= det_loc[i].w &&
                        absf(p.y - det_loc[i].y) <= det_loc[i].w &&
                        absf(p.z - det_loc[i].z) <= det_loc[i].w )
                        gpu_set(g.det_hit, photon_idx, i);
            }
        }
    }
}

// Make sure the source is at an interface.
void correct_source(Simulation *sim)
{
    uint8_t media_index;
    int3 p;
    float3 r0;

    // Source's position (euclidean).
    r0.x = sim->src.r.x; r0.y = sim->src.r.y; r0.z = sim->src.r.z;

    MOVE(p, r0, sim->grid.stepr);

    media_index = sim->grid.media_type[p.x][p.y][p.z];

    while( media_index != 0 &&
           p.x > 0 && p.x < sim->grid.dim.x &&
           p.y > 0 && p.y < sim->grid.dim.y &&
           p.z > 0 && p.z < sim->grid.dim.z )
    {
        r0.x -= sim->src.d.x * sim->grid.minstepsize;
        r0.y -= sim->src.d.y * sim->grid.minstepsize;
        r0.z -= sim->src.d.z * sim->grid.minstepsize;
        MOVE(p, r0, sim->grid.stepr);
        media_index = sim->grid.media_type[p.x][p.y][p.z];
    }
    while( media_index == 0 &&
           p.x > 0 && p.x < sim->grid.dim.x &&
           p.y > 0 && p.y < sim->grid.dim.y &&
           p.z > 0 && p.z < sim->grid.dim.z )
    {
        r0.x += sim->src.d.x * sim->grid.minstepsize;
        r0.y += sim->src.d.y * sim->grid.minstepsize;
        r0.z += sim->src.d.z * sim->grid.minstepsize;
        MOVE(p, r0, sim->grid.stepr);
        media_index = sim->grid.media_type[p.x][p.y][p.z];
    }

    // Update the source coordinates 
    sim->src.r.x = r0.x;
    sim->src.r.y = r0.y;
    sim->src.r.z = r0.z;
}

void simulate(ExecConfig conf, Simulation sim, GPUMemory gmem)
{
    uint32_t seed;
    uint32_t *temp_seed, *d_seed;
    int photons_per_iteration = sim.n_photons / conf.n_iterations;
    int photons_per_thread = photons_per_iteration / conf.n_threads;
    int iteration = 0;

    printf("photons per thread = %d\n", photons_per_thread);
    printf("photons per iteration = %d\n", photons_per_iteration);

    seed = conf.rand_seed;
    d_seed = gmem.seed;
    for(iteration = 0; iteration < conf.n_iterations; iteration++)
    {
        run_simulation<<< conf.n_blocks, 128 >>>(d_seed, photons_per_thread, iteration);

        // Order a new batch of RNG seeds while the current iteration is being simulated.
        temp_seed = init_rand_seed(seed++, conf);

        // Make sure all photons have already been simulated before moving on.
        hipDeviceSynchronize();

        hipFree(d_seed);
        d_seed = temp_seed;
    }
}
