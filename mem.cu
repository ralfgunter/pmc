#include "hip/hip_runtime.h"
/********************************************************************************
*          Monte-Carlo Simulation for Light Transport in 3D Volumes             *
*********************************************************************************
*                                                                               *
* Copyright (C) 2002-2008,  David Boas    (dboas <at> nmr.mgh.harvard.edu)      *
*               2008        Jay Dubb      (jdubb <at> nmr.mgh.harvard.edu)      *
*               2008        Qianqian Fang (fangq <at> nmr.mgh.harvard.edu)      *
*               2011        Ralf Gunter   (ralfgunter <at> gmail.com)           *
*                                                                               *
* License:  4-clause BSD License, see LICENSE for details                       *
*                                                                               *
********************************************************************************/

#include "main.h"

template <class type>
void linearize_3d(type ***t, type *l, int dim_x, int dim_y, int dim_z)
{
    for (int x = 0; x < dim_x; x++)
        for (int y = 0; y < dim_y; y++)
            for (int z = 0; z < dim_z; z++)
                l[LIN3D(x,y,z,dim_x,dim_y)] = t[x][y][z];
}

uint32_t* init_rand_seed(int seed, ExecConfig conf)
{
    uint32_t *h_seed, *d_seed;
    size_t sizeof_seed;

    if(seed > 0)
        srand(seed);
    else
        srand(time(NULL));

    // Seed used by the RNG.
    sizeof_seed = conf.n_threads * RAND_SEED_LEN * sizeof(uint32_t);
    h_seed = (uint32_t *) malloc(sizeof_seed);
    for(int i = 0; i < conf.n_threads * RAND_SEED_LEN; i++)
        h_seed[i] = rand();

    hipMalloc((void **) &d_seed, sizeof_seed);
    TO_DEVICE(d_seed, h_seed, sizeof_seed);

    free(h_seed);

    return d_seed;
}

void init_mem(ExecConfig conf, Simulation *sim, GPUMemory *gmem)
{
    float *d_fbox;
    float *d_path_length, *d_mom_transfer;
    uint8_t *h_linear_media_type, *d_media_type;
    int8_t *d_det_hit;
    uint32_t *d_seed;
    int4 *d_det_loc;
    float4 *d_media_prop;
    size_t num_tissueArrays, num_fbox;

    // Calculate the total number of voxel elements.
    int grid_dim = sim->grid.dim.x * sim->grid.dim.y * sim->grid.dim.z;

    // Linearize media_type, as CUDA cannot handle pointers to pointers.
    h_linear_media_type = (uint8_t *) malloc(grid_dim * sizeof(uint8_t));
    linearize_3d(sim->grid.media_type, h_linear_media_type,
                 sim->grid.dim.x, sim->grid.dim.y, sim->grid.dim.z);

    // Setup the path length and momentum transfer arrays.
    //num_tissueArrays = (sim->tiss.num + 1) * sim->n_photons;
    num_tissueArrays = 1 << NUM_HASH_BITS; // 128 MBs used by each array; must be a power of 2
    sim->path_length  = (float *) calloc(num_tissueArrays, sizeof(float));
    sim->mom_transfer = (float *) calloc(num_tissueArrays, sizeof(float));

    // Photon fluence.
    num_fbox = sim->grid.nIxyz * sim->num_time_steps;
    sim->fbox = (float *) calloc(num_fbox, sizeof(float));

    // Array of which photons hit which detectors (if any).
    sim->det.hit = (int8_t *) calloc(sim->n_photons, sizeof(int8_t));

    // Array of seeds for the GPU random number generator.
    d_seed = init_rand_seed(conf.rand_seed, conf);

    // Allocate memory on the GPU global memory.
    hipMalloc((void **) &d_det_loc, MAX_DETECTORS * sizeof(int4));
    hipMalloc((void **) &d_media_prop, (MAX_TISSUES + 1) * sizeof(float4));
    hipMalloc((void **) &d_media_type, grid_dim * sizeof(uint8_t));
    hipMalloc((void **) &d_path_length,  num_tissueArrays * sizeof(float));
    hipMalloc((void **) &d_mom_transfer, num_tissueArrays * sizeof(float));
    hipMalloc((void **) &d_fbox,         num_fbox         * sizeof(float));
    hipMalloc((void **) &d_det_hit, sim->n_photons * sizeof(int8_t));

    printf("det_loc: %ld\n", MAX_DETECTORS * sizeof(int4));
    printf("media_prop: %ld\n", (MAX_TISSUES + 1) * sizeof(float4));
    printf("media_type: %ld\n", grid_dim * sizeof(uint8_t));
    printf("path_length: %ld\n",  num_tissueArrays * sizeof(float));
    printf("mom_transfer: %ld\n", num_tissueArrays * sizeof(float));
    printf("fbox: %ld\n",         num_fbox         * sizeof(float));
    printf("det_hit: %ld\n", sim->n_photons * sizeof(int8_t));

    int gpu_mem_spent = sizeof(int4) * MAX_DETECTORS
                      + sizeof(float4) * (MAX_TISSUES + 1)
                      + sizeof(uint8_t) * grid_dim
                      + sizeof(float) * num_tissueArrays
                      + sizeof(float) * num_tissueArrays
                      + sizeof(float) * num_fbox
                      + sizeof(int8_t) * sim->n_photons
                      + sizeof(uint32_t) * conf.n_threads * RAND_SEED_LEN;
    printf("memory spent = %dMB\n", gpu_mem_spent / (1024 * 1024));

    // Copy simulation memory to the GPU.
    //hipMemcpyToSymbol(HIP_SYMBOL("det_loc"), sim->det.info, sim->det.num * sizeof(int4));
    //hipMemcpyToSymbol(HIP_SYMBOL("media_prop"), sim->tiss.prop, (sim->tiss.num + 1) * sizeof(float4));
    hipMemcpyToSymbol(HIP_SYMBOL("s"), sim, sizeof(Simulation));
    TO_DEVICE(d_det_loc, sim->det.info, MAX_DETECTORS * sizeof(int4));
    TO_DEVICE(d_media_prop, sim->tiss.prop, (MAX_TISSUES + 1) * sizeof(float4));
    TO_DEVICE(d_media_type, h_linear_media_type, grid_dim * sizeof(uint8_t));
    TO_DEVICE(d_path_length, sim->path_length, num_tissueArrays * sizeof(float));
    TO_DEVICE(d_mom_transfer, sim->mom_transfer, num_tissueArrays * sizeof(float));
    TO_DEVICE(d_fbox,      sim->fbox,      num_fbox           * sizeof(float));
    TO_DEVICE(d_det_hit, sim->det.hit, sim->n_photons * sizeof(int8_t));

    // Update GPU memory structure (so that its pointers can be used elsewhere).
    gmem->det_loc = d_det_loc;
    gmem->media_prop = d_media_prop;
    gmem->media_type = d_media_type;
    gmem->path_length = d_path_length;
    gmem->mom_transfer = d_mom_transfer;
    gmem->fbox = d_fbox;
    gmem->det_hit = d_det_hit;
    gmem->seed = d_seed;
    hipMemcpyToSymbol(HIP_SYMBOL("g"), gmem, sizeof(GPUMemory));

    // Free temporary memory used on the host.
    free(h_linear_media_type);
}

void free_gpu_results_mem(GPUMemory gmem)
{
    // Path length and momentum transfer.
    hipFree(gmem.path_length);
    hipFree(gmem.mom_transfer);

    // Photon fluence.
    hipFree(gmem.fbox);

    hipFree(gmem.det_hit);
}

void free_gpu_params_mem(GPUMemory gmem)
{
    // Tissue types.
    hipFree(gmem.media_type);

    // Detectors' locations and radii.
    hipFree(gmem.det_loc);

    // Optical properties of the different tissue types.
    hipFree(gmem.media_prop);

    // Random number generation.
    hipFree(gmem.seed);
}

void free_cpu_results_mem(Simulation sim)
{
    // Path length and momentum transfer.
    free(sim.path_length);
    free(sim.mom_transfer);

    // Photon fluence.
    free(sim.fbox);

    free(sim.det.hit);
}

void free_cpu_params_mem(Simulation sim)
{
    // Tissue types.
    for(int i = 0; i < sim.grid.dim.x; i++) {
        for(int j = 0; j < sim.grid.dim.y; j++) {
            free(sim.grid.media_type[i][j]);
        }
        free(sim.grid.media_type[i]);
    }
    free(sim.grid.media_type);

    // Detectors' locations and radii.
    free(sim.det.info);

    // Optical properties of the different tissue types.
    free(sim.tiss.prop);
}

void free_mem(Simulation sim, GPUMemory gmem)
{
    free_gpu_params_mem(gmem); free_gpu_results_mem(gmem);
    free_cpu_params_mem(sim);  free_cpu_results_mem(sim);
}

void retrieve(Simulation *sim, GPUMemory *gmem)
{
    //size_t sizeof_tissueArrays = sim->n_photons * (sim->tiss.num + 1) * sizeof(float);
    size_t sizeof_tissueArrays = (1 << NUM_HASH_BITS) * sizeof(float);
    size_t sizeof_fbox = sim->grid.nIxyz * sim->num_time_steps * sizeof(float);
    size_t sizeof_det_hit = sim->n_photons * sizeof(int8_t);

    TO_HOST(sim->path_length, gmem->path_length, sizeof_tissueArrays);
    TO_HOST(sim->mom_transfer, gmem->mom_transfer, sizeof_tissueArrays);
    TO_HOST(sim->fbox, gmem->fbox, sizeof_fbox);
    TO_HOST(sim->det.hit, gmem->det_hit, sizeof_det_hit);
}
