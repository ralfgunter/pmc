#include "hip/hip_runtime.h"
/********************************************************************************
*          Monte-Carlo Simulation for Light Transport in 3D Volumes             *
*********************************************************************************
*                                                                               *
* Copyright (C) 2002-2008,  David Boas    (dboas <at> nmr.mgh.harvard.edu)      *
*               2008        Jay Dubb      (jdubb <at> nmr.mgh.harvard.edu)      *
*               2008        Qianqian Fang (fangq <at> nmr.mgh.harvard.edu)      *
*               2011        Ralf Gunter   (ralfgunter <at> gmail.com)           *
*                                                                               *
* License:  4-clause BSD License, see LICENSE for details                       *
*                                                                               *
* Example:                                                                      *
*         tMCimg input.inp                                                      *
*                                                                               *
* Please find more details in README and doc/HELP                               *
********************************************************************************/

#include "main.h"

template <class type>
void linearize_3d(type ***t, type *l, int dim_x, int dim_y, int dim_z)
{
    for (int x = 0; x < dim_x; x++)
        for (int y = 0; y < dim_y; y++)
            for (int z = 0; z < dim_z; z++)
                l[LIN3D(x,y,z,dim_x,dim_y)] = t[x][y][z];
}

void init_mem(ExecConfig conf, Simulation *sim, GPUMemory *gmem)
{
    Real *d_II;
    Real *d_lenTiss, *d_momTiss;
    uint *d_detHit_matrix;
    uint *h_seed, *d_seed;
    uchar *h_linear_tissueType, *d_tissueType;
    size_t num_tissueArrays, num_II, sizeof_seed;

    // Total number of voxel elements
    int grid_dim = sim->grid.dim.x * sim->grid.dim.y * sim->grid.dim.z;

    // Linearize tissueType so that it can be converted into a hipArray
    // and later bound to a 3d texture.
    h_linear_tissueType = (uchar *) malloc(grid_dim * sizeof(uchar));
    linearize_3d(sim->grid.tissueType, h_linear_tissueType,
                 sim->grid.dim.x, sim->grid.dim.y, sim->grid.dim.z);

    // Setup the path length and momentum transfer arrays.
    num_tissueArrays = (sim->tiss.num + 1) * sim->n_photons;
    sim->lenTiss = (Real *) calloc(num_tissueArrays, sizeof(Real));
    sim->momTiss = (Real *) calloc(num_tissueArrays, sizeof(Real));

    // Photon fluence.
    num_II = sim->grid.nIxyz * sim->max_time;
    sim->II = (Real *) calloc(num_II, sizeof(Real));

    // Bitset indicating which detectors (if any) were hit by which photons.
    sim->detHit = bitset_new(sim->n_photons, sim->det.num);

    // Seed used by the RNG.
    sizeof_seed = sizeof(uint) * conf.n_threads * RAND_SEED_LEN;
    h_seed = (uint *) malloc(sizeof_seed);
    for(int i = 0; i < conf.n_threads * RAND_SEED_LEN; i++)
        h_seed[i] = rand();

    // Allocate memory on the GPU global memory.
    // TODO: use constant memory where appropriate 
    hipMalloc((void **) &d_tissueType, grid_dim * sizeof(uchar));
    hipMalloc((void **) &d_lenTiss,  num_tissueArrays * sizeof(Real));
    hipMalloc((void **) &d_momTiss,  num_tissueArrays * sizeof(Real));
    hipMalloc((void **) &d_II,       num_II           * sizeof(Real));
    hipMalloc((void **) &d_detHit_matrix, bitset_size(sim->detHit) * sizeof(uint));
    hipMalloc((void **) &d_seed, sizeof_seed);

    // Copy simulation memory to the GPU.
    hipMemcpyToSymbol(HIP_SYMBOL("detLoc"), sim->det.info, sim->det.num * sizeof(int4));
    hipMemcpyToSymbol(HIP_SYMBOL("tissueProp"), sim->tiss.prop, (sim->tiss.num + 1) * sizeof(float4));
    TO_DEVICE(d_tissueType, h_linear_tissueType, grid_dim * sizeof(uchar));
    TO_DEVICE(d_lenTiss, sim->lenTiss,      num_tissueArrays * sizeof(Real));
    TO_DEVICE(d_momTiss, sim->momTiss,      num_tissueArrays * sizeof(Real));
    TO_DEVICE(d_II,      sim->II,           num_II           * sizeof(Real));
    TO_DEVICE(d_detHit_matrix, sim->detHit.matrix, bitset_size(sim->detHit) * sizeof(uint));
    TO_DEVICE(d_seed, h_seed, sizeof_seed);

    // Update GPU memory structure (so that its pointers can be used elsewhere).
    gmem->tissueType = d_tissueType;
    gmem->lenTiss = d_lenTiss;
    gmem->momTiss = d_momTiss;
    gmem->II = d_II;
    gmem->detHit = sim->detHit;
    gmem->detHit.matrix = d_detHit_matrix;
    gmem->seed = d_seed;

    // Free temporary memory used on the host.
    free(h_linear_tissueType);
    free(h_seed);
}

void free_mem(Simulation sim, GPUMemory gmem)
{
    // Tissue types.
    for(int i = 0; i < sim.grid.dim.x; i++) {
        for(int j = 0; j < sim.grid.dim.y; j++) {
            free(sim.grid.tissueType[i][j]);
        }
        free(sim.grid.tissueType[i]);
    }
    free(sim.grid.tissueType);
    hipFree(gmem.tissueType);

    // Detectors' locations and radii.
    free(sim.det.info);

    // Optical properties of the different tissue types.
    free(sim.tiss.prop);

    // Path length and momentum transfer.
    free(sim.lenTiss);
    free(sim.momTiss);
    hipFree(gmem.lenTiss);
    hipFree(gmem.momTiss);

    // Photon fluence.
    free(sim.II);
    hipFree(gmem.II);

    // Random number generation.
    hipFree(gmem.seed);

    // Bitset of the detectors which were hit by a given photon.
    bitset_free(sim.detHit);
    hipFree(gmem.detHit.matrix);  // TODO: properly handle this 
}

void retrieve(Simulation *sim, GPUMemory *gmem)
{
    size_t sizeof_tissueArrays = sim->n_photons * (sim->tiss.num + 1) * sizeof(Real);
    size_t sizeof_II = sim->grid.nIxyz * sim->max_time * sizeof(Real);
    size_t sizeof_detHit = bitset_size(sim->detHit) * sizeof(uint);

    TO_HOST(sim->lenTiss, gmem->lenTiss, sizeof_tissueArrays);
    TO_HOST(sim->momTiss, gmem->momTiss, sizeof_tissueArrays);
    TO_HOST(sim->II, gmem->II, sizeof_II);
    TO_HOST(sim->detHit.matrix, gmem->detHit.matrix, sizeof_detHit);
}
