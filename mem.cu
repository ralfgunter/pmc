#include "hip/hip_runtime.h"
/********************************************************************************
*          Monte-Carlo Simulation for Light Transport in 3D Volumes             *
*********************************************************************************
*                                                                               *
* Copyright (C) 2002-2008,  David Boas    (dboas <at> nmr.mgh.harvard.edu)      *
*               2008        Jay Dubb      (jdubb <at> nmr.mgh.harvard.edu)      *
*               2008        Qianqian Fang (fangq <at> nmr.mgh.harvard.edu)      *
*               2011        Ralf Gunter   (ralfgunter <at> gmail.com)           *
*                                                                               *
* License:  4-clause BSD License, see LICENSE for details                       *
*                                                                               *
********************************************************************************/

#include "main.h"

template <class type>
void linearize_3d(type ***t, type *l, int dim_x, int dim_y, int dim_z)
{
    for (int x = 0; x < dim_x; x++)
        for (int y = 0; y < dim_y; y++)
            for (int z = 0; z < dim_z; z++)
                l[LIN3D(x,y,z,dim_x,dim_y)] = t[x][y][z];
}

uint32_t* init_rand_seed(uint32_t seed, ExecConfig conf)
{
    uint32_t *h_seed, *d_seed;
    size_t sizeof_seed;

    if(seed > 0)
        srand(seed);
    else
        srand(time(NULL));

    // Seed used by the RNG.
    sizeof_seed = conf.n_threads * RAND_SEED_LEN * sizeof(uint32_t);
    h_seed = (uint32_t *) malloc(sizeof_seed);
    for(int i = 0; i < conf.n_threads * RAND_SEED_LEN; i++)
        h_seed[i] = rand();

    hipMalloc((void **) &d_seed, sizeof_seed);
    TO_DEVICE(d_seed, h_seed, sizeof_seed);

    free(h_seed);

    return d_seed;
}

void init_mem(ExecConfig conf, Simulation *sim, GPUMemory *gmem)
{
    float *d_II;
    float *d_lenTiss, *d_momTiss;
    uint8_t *h_linear_tissueType, *d_tissueType;
    uint32_t *d_detHit_matrix;
    uint32_t *d_seed;
    int4 *d_detLoc;
    float4 *d_tissueProp;
    size_t num_tissueArrays, num_II;

    // Calculate the total number of voxel elements.
    int grid_dim = sim->grid.dim.x * sim->grid.dim.y * sim->grid.dim.z;

    // Linearize tissueType, as CUDA cannot handle pointers to pointers.
    h_linear_tissueType = (uint8_t *) malloc(grid_dim * sizeof(uint8_t));
    linearize_3d(sim->grid.tissueType, h_linear_tissueType,
                 sim->grid.dim.x, sim->grid.dim.y, sim->grid.dim.z);

    // Setup the path length and momentum transfer arrays.
    //num_tissueArrays = (sim->tiss.num + 1) * sim->n_photons;
    num_tissueArrays = 1 << NUM_HASH_BITS; // 128 MBs used by each array; must be a power of 2
    sim->lenTiss = (float *) calloc(num_tissueArrays, sizeof(float));
    sim->momTiss = (float *) calloc(num_tissueArrays, sizeof(float));

    // Photon fluence.
    num_II = sim->grid.nIxyz * sim->max_time;
    sim->II = (float *) calloc(num_II, sizeof(float));

    // Bitset indicating which detectors (if any) were hit by which photons.
    sim->detHit = bitset_new(sim->n_photons, sim->det.num);

    d_seed = init_rand_seed(conf.rand_seed, conf);

    // Allocate memory on the GPU global memory.
    // TODO: use constant memory where appropriate 
    hipMalloc((void **) &d_detLoc, sim->det.num * sizeof(int4));
    hipMalloc((void **) &d_tissueProp, (sim->tiss.num + 1) * sizeof(float4));
    hipMalloc((void **) &d_tissueType, grid_dim * sizeof(uint8_t));
    hipMalloc((void **) &d_lenTiss, num_tissueArrays * sizeof(float));
    hipMalloc((void **) &d_momTiss, num_tissueArrays * sizeof(float));
    hipMalloc((void **) &d_II,      num_II           * sizeof(float));
    hipMalloc((void **) &d_detHit_matrix, bitset_size(sim->detHit) * sizeof(uint32_t));

    int gpu_mem_spent = sizeof(int4)   * sim->det.num
                      + sizeof(float4) * (sim->tiss.num + 1)
                      + sizeof(uint8_t)  * grid_dim
                      + sizeof(float)  * num_tissueArrays
                      + sizeof(float)  * num_tissueArrays
                      + sizeof(float)  * num_II
                      + sizeof(uint32_t)   * bitset_size(sim->detHit)
                      + sizeof(uint32_t)   * conf.n_threads * RAND_SEED_LEN;
    printf("memory spent = %dMB\n", gpu_mem_spent / (1024 * 1024));

    // Copy simulation memory to the GPU.
    //hipMemcpyToSymbol(HIP_SYMBOL("detLoc"), sim->det.info, sim->det.num * sizeof(int4));
    //hipMemcpyToSymbol(HIP_SYMBOL("tissueProp"), sim->tiss.prop, (sim->tiss.num + 1) * sizeof(float4));
    hipMemcpyToSymbol(HIP_SYMBOL("s"), sim, sizeof(Simulation));
    TO_DEVICE(d_detLoc, sim->det.info, sim->det.num * sizeof(int4));
    TO_DEVICE(d_tissueProp, sim->tiss.prop, (sim->tiss.num + 1) * sizeof(float4));
    TO_DEVICE(d_tissueType, h_linear_tissueType, grid_dim * sizeof(uint8_t));
    TO_DEVICE(d_lenTiss, sim->lenTiss, num_tissueArrays * sizeof(float));
    TO_DEVICE(d_momTiss, sim->momTiss, num_tissueArrays * sizeof(float));
    TO_DEVICE(d_II,      sim->II,      num_II           * sizeof(float));
    TO_DEVICE(d_detHit_matrix, sim->detHit.matrix, bitset_size(sim->detHit) * sizeof(uint32_t));

    // Update GPU memory structure (so that its pointers can be used elsewhere).
    gmem->detLoc = d_detLoc;
    gmem->tissueProp = d_tissueProp;
    gmem->tissueType = d_tissueType;
    gmem->lenTiss = d_lenTiss;
    gmem->momTiss = d_momTiss;
    gmem->II = d_II;
    gmem->detHit = sim->detHit;
    gmem->detHit.matrix = d_detHit_matrix;
    gmem->seed = d_seed;
    hipMemcpyToSymbol(HIP_SYMBOL("g"), gmem, sizeof(GPUMemory));

    // Free temporary memory used on the host.
    free(h_linear_tissueType);
}

void free_gpu_mem(GPUMemory gmem)
{
    // Tissue types.
    hipFree(gmem.tissueType);

    // Detectors' locations and radii.
    hipFree(gmem.detLoc);

    // Optical properties of the different tissue types.
    hipFree(gmem.tissueProp);

    // Path length and momentum transfer.
    hipFree(gmem.lenTiss);
    hipFree(gmem.momTiss);

    // Photon fluence.
    hipFree(gmem.II);

    // Bitset of the detectors which were hit by a given photon.
    hipFree(gmem.detHit.matrix);  // TODO: properly handle this 

    // Random number generation.
    hipFree(gmem.seed);
}

void free_cpu_mem(Simulation sim)
{
    // Tissue types.
    for(int i = 0; i < sim.grid.dim.x; i++) {
        for(int j = 0; j < sim.grid.dim.y; j++) {
            free(sim.grid.tissueType[i][j]);
        }
        free(sim.grid.tissueType[i]);
    }
    free(sim.grid.tissueType);

    // Detectors' locations and radii.
    free(sim.det.info);

    // Optical properties of the different tissue types.
    free(sim.tiss.prop);

    // Path length and momentum transfer.
    free(sim.lenTiss);
    free(sim.momTiss);

    // Photon fluence.
    free(sim.II);

    // Bitset of the detectors which were hit by a given photon.
    bitset_free(sim.detHit);
}

void free_mem(Simulation sim, GPUMemory gmem)
{
    free_gpu_mem(gmem); free_cpu_mem(sim);
}

void retrieve(Simulation *sim, GPUMemory *gmem)
{
    //size_t sizeof_tissueArrays = sim->n_photons * (sim->tiss.num + 1) * sizeof(float);
    size_t sizeof_tissueArrays = (1 << NUM_HASH_BITS) * sizeof(float);
    size_t sizeof_II = sim->grid.nIxyz * sim->max_time * sizeof(float);
    size_t sizeof_detHit = bitset_size(sim->detHit) * sizeof(uint32_t);

    TO_HOST(sim->lenTiss, gmem->lenTiss, sizeof_tissueArrays);
    TO_HOST(sim->momTiss, gmem->momTiss, sizeof_tissueArrays);
    TO_HOST(sim->II, gmem->II, sizeof_II);
    TO_HOST(sim->detHit.matrix, gmem->detHit.matrix, sizeof_detHit);
}
