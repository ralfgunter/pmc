#include "hip/hip_runtime.h"
/********************************************************************************
*          Monte-Carlo Simulation for Light Transport in 3D Volumes             *
*********************************************************************************
*                                                                               *
* Copyright (C) 2002-2008,  David Boas    (dboas <at> nmr.mgh.harvard.edu)      *
*               2008        Jay Dubb      (jdubb <at> nmr.mgh.harvard.edu)      *
*               2008        Qianqian Fang (fangq <at> nmr.mgh.harvard.edu)      *
*               2011        Ralf Gunter   (ralfgunter <at> gmail.com)           *
*                                                                               *
* License:  4-clause BSD License, see LICENSE for details                       *
*                                                                               *
********************************************************************************/

#include "main.h"

template <class type>
void linearize_3d(type ***t, type *l, int dim_x, int dim_y, int dim_z)
{
    for (int x = 0; x < dim_x; x++)
        for (int y = 0; y < dim_y; y++)
            for (int z = 0; z < dim_z; z++)
                l[LIN3D(x,y,z,dim_x,dim_y)] = t[x][y][z];
}

void init_mem(ExecConfig conf, Simulation *sim, GPUMemory *gmem)
{
    float *d_II;
    //float *d_lenTiss, *d_momTiss;
    //uint *d_detHit_matrix;
    uint *h_seed, *d_seed;
    uchar *h_linear_tissueType, *d_tissueType;
    //int4 *d_detLoc;
    float4 *d_tissueProp;
    size_t num_tissueArrays, num_II, sizeof_seed;

    // Calculate the total number of voxel elements.
    int grid_dim = sim->grid.dim.x * sim->grid.dim.y * sim->grid.dim.z;

    // Linearize tissueType so that it can be converted into a hipArray
    // and later bound to a 3d texture.
    h_linear_tissueType = (uchar *) malloc(grid_dim * sizeof(uchar));
    linearize_3d(sim->grid.tissueType, h_linear_tissueType,
                 sim->grid.dim.x, sim->grid.dim.y, sim->grid.dim.z);

/*
    hipArray *d_volumeArray = 0;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    const hipExtent volumeSize = make_hipExtent(sim->grid.dim.x,
                                                  sim->grid.dim.y,
                                                  sim->grid.dim.z);
    cutilSafeCall( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)h_linear_tissueType, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );

    // set texture parameters
    tissueType.normalized = false;   // access with normalized texture coordinates
    tissueType.filterMode = hipFilterModeLinear;      // linear interpolation
    tissueType.addressMode[0] = hipAddressModeWrap;   // wrap tissueTypeture coordinates
    tissueType.addressMode[1] = hipAddressModeWrap;
    tissueType.addressMode[2] = hipAddressModeWrap;

    // bind array to 3D texture
    cutilSafeCall( hipBindTextureToArray(tissueType, d_volumeArray, channelDesc) );
*/

    // Setup the path length and momentum transfer arrays.
    //num_tissueArrays = (sim->tiss.num + 1) * sim->n_photons;
    //sim->lenTiss = (float *) calloc(num_tissueArrays, sizeof(float));
    //sim->momTiss = (float *) calloc(num_tissueArrays, sizeof(float));

    // Photon fluence.
    num_II = sim->grid.nIxyz * sim->max_time;
    sim->II = (float *) calloc(num_II, sizeof(float));

    // Bitset indicating which detectors (if any) were hit by which photons.
    //sim->detHit = bitset_new(sim->n_photons, sim->det.num);

    // Seed used by the RNG.
    sizeof_seed = conf.n_threads * RAND_SEED_LEN * sizeof(uint);
    h_seed = (uint *) malloc(sizeof_seed);
    for(int i = 0; i < conf.n_threads * RAND_SEED_LEN; i++)
        h_seed[i] = rand();

    // Allocate memory on the GPU global memory.
    // TODO: use constant memory where appropriate 
    //hipMalloc((void **) &d_detLoc, sim->det.num * sizeof(int4));
    hipMalloc((void **) &d_tissueProp, (sim->tiss.num + 1) * sizeof(float4));
    hipMalloc((void **) &d_tissueType, grid_dim * sizeof(uchar));
    //hipMalloc((void **) &d_lenTiss, num_tissueArrays * sizeof(float));
    //hipMalloc((void **) &d_momTiss, num_tissueArrays * sizeof(float));
    hipMalloc((void **) &d_II,      num_II           * sizeof(float));
    //hipMalloc((void **) &d_detHit_matrix, bitset_size(sim->detHit) * sizeof(uint));
    hipMalloc((void **) &d_seed, sizeof_seed);

    // Copy simulation memory to the GPU.
    //hipMemcpyToSymbol(HIP_SYMBOL("detLoc"), sim->det.info, sim->det.num * sizeof(int4));
    //hipMemcpyToSymbol(HIP_SYMBOL("tissueProp"), sim->tiss.prop, (sim->tiss.num + 1) * sizeof(float4));
    hipMemcpyToSymbol(HIP_SYMBOL("s"), sim, sizeof(Simulation));
    //TO_DEVICE(d_detLoc, sim->det.info, sim->det.num * sizeof(int4));
    TO_DEVICE(d_tissueProp, sim->tiss.prop, (sim->tiss.num + 1) * sizeof(float4));
    TO_DEVICE(d_tissueType, h_linear_tissueType, grid_dim * sizeof(uchar));
    //TO_DEVICE(d_lenTiss, sim->lenTiss, num_tissueArrays * sizeof(float));
    //TO_DEVICE(d_momTiss, sim->momTiss, num_tissueArrays * sizeof(float));
    TO_DEVICE(d_II,      sim->II,      num_II           * sizeof(float));
    //TO_DEVICE(d_detHit_matrix, sim->detHit.matrix, bitset_size(sim->detHit) * sizeof(uint));
    TO_DEVICE(d_seed, h_seed, sizeof_seed);

    // Update GPU memory structure (so that its pointers can be used elsewhere).
    //gmem->detLoc = d_detLoc;
    gmem->tissueProp = d_tissueProp;
    gmem->tissueType = d_tissueType;
    //gmem->lenTiss = d_lenTiss;
    //gmem->momTiss = d_momTiss;
    gmem->II = d_II;
    //gmem->detHit = sim->detHit;
    //gmem->detHit.matrix = d_detHit_matrix;
    gmem->seed = d_seed;
    hipMemcpyToSymbol(HIP_SYMBOL("g"), gmem, sizeof(GPUMemory));

    // Free temporary memory used on the host.
    free(h_linear_tissueType);
    free(h_seed);
}

void free_mem(Simulation sim, GPUMemory gmem)
{
    // Tissue types.
    for(int i = 0; i < sim.grid.dim.x; i++) {
        for(int j = 0; j < sim.grid.dim.y; j++) {
            free(sim.grid.tissueType[i][j]);
        }
        free(sim.grid.tissueType[i]);
    }
    free(sim.grid.tissueType);
    hipFree(gmem.tissueType);
    //hipUnbindTexture(tissueType);

    // Detectors' locations and radii.
    //free(sim.det.info);
    //hipFree(gmem.detLoc);

    // Optical properties of the different tissue types.
    free(sim.tiss.prop);
    hipFree(gmem.tissueProp);

    // Path length and momentum transfer.
    //free(sim.lenTiss);
    //free(sim.momTiss);
    //hipFree(gmem.lenTiss);
    //hipFree(gmem.momTiss);

    // Photon fluence.
    free(sim.II);
    hipFree(gmem.II);

    // Random number generation.
    hipFree(gmem.seed);

    // Bitset of the detectors which were hit by a given photon.
    //bitset_free(sim.detHit);
    //hipFree(gmem.detHit.matrix);  // TODO: properly handle this 
}

void retrieve(Simulation *sim, GPUMemory *gmem)
{
    size_t sizeof_tissueArrays = sim->n_photons * (sim->tiss.num + 1) * sizeof(float);
    size_t sizeof_II = sim->grid.nIxyz * sim->max_time * sizeof(float);
    size_t sizeof_detHit = bitset_size(sim->detHit) * sizeof(uint);

    //TO_HOST(sim->lenTiss, gmem->lenTiss, sizeof_tissueArrays);
    //TO_HOST(sim->momTiss, gmem->momTiss, sizeof_tissueArrays);
    TO_HOST(sim->II, gmem->II, sizeof_II);
    TO_HOST(sim->detHit.matrix, gmem->detHit.matrix, sizeof_detHit);
}
