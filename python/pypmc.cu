#include "hip/hip_runtime.h"
#include <Python.h>
#include "structmember.h"
#include "main.h"

typedef struct {
    PyObject_HEAD

    PyObject *py_pathlength, *py_momentum_transfer;
    PyObject *py_medium;

    ExecConfig conf;
    Simulation sim;
    GPUMemory gmem;
} PyPMC;

static PyObject* pypmc_get_tissueArray( Simulation sim, float *tissueArray );

////////////////////////////////////////////////////////////////////
//// Fundamental methods
static void
pypmc_dealloc( PyPMC *self )
{
    // Call PMC's own cleaning up procedure.
    free_mem(self->sim, self->gmem);

    Py_XDECREF(self->py_pathlength);
    Py_XDECREF(self->py_momentum_transfer);

    // Finally, delete the pypmc object itself.
    Py_TYPE(self)->tp_free((PyObject*) self);
}

static PyObject *
pypmc_new( PyTypeObject *type, PyObject *args, PyObject *kwds )
{
    PyPMC *self = (PyPMC *) type->tp_alloc(type, 0);

    return (PyObject *) self;
}

static int
pypmc_init( PyPMC *self, PyObject *args )
{
    const char *input_filepath;
    int n_threads, n_iterations;

    // The user may optionally pass the .inp filepath to automatically load
    // the simulation parameters from there.
    if(! PyArg_ParseTuple(args, "|sii", &input_filepath, &n_threads, &n_iterations))
        return -1;

    if(input_filepath == NULL)
        return 0;

    // Parse .inp file into the simulation structure.
    read_input(&self->conf, &self->sim, input_filepath);

    parse_conf(&self->conf, n_threads, n_iterations);

    // Make sure the source is at an interface.
    correct_source(&self->sim);

    // Allocate and initialize memory to be used by the GPU.
    init_mem(self->conf, &self->sim, &self->gmem);

    return 0;
}

//// end of fundamental methods
//////////////////////////////////////////////////////////////////////////////

static PyObject *
pypmc_write_to_disk( PyPMC *self, PyObject *args )
{
    const char *output_filepath;    

    if (! PyArg_ParseTuple(args, "s", &output_filepath))
        return NULL;

    write_results(self->sim, output_filepath);

    Py_RETURN_NONE;
}

static PyObject *
pypmc_run( PyPMC *self, PyObject *args )
{
    // Run simulations on the GPU.
    simulate(self->conf, self->sim, self->gmem);

    Py_RETURN_NONE;
}

static PyObject *
pypmc_pull_results( PyPMC *self, PyObject *args )
{
    // Retrieve results to host.
    retrieve(&self->sim, &self->gmem);

    self->py_pathlength = pypmc_get_tissueArray(self->sim, self->sim.lenTiss);
    self->py_momentum_transfer = pypmc_get_tissueArray(self->sim, self->sim.momTiss);

    Py_RETURN_NONE;
}

static PyObject *
pypmc_push_parameters( PyPMC *self, PyObject *args )
{
    // Allocate and initialize memory to be used by the GPU.
    free_gpu_mem(self->gmem);
    init_mem(self->conf, &self->sim, &self->gmem);

    Py_RETURN_NONE;
}

static PyObject *
pypmc_load_medium( PyPMC *self, PyObject *args )
{
    const char *medium_filepath;
    int dim_x, dim_y, dim_z;

    if (! PyArg_ParseTuple(args, "siii", &medium_filepath, &dim_x, &dim_y, &dim_z))
        return NULL;

    self->sim.grid.dim.x = dim_x;
    self->sim.grid.dim.y = dim_y;
    self->sim.grid.dim.z = dim_z;

    read_segmentation_file(&self->sim, medium_filepath);

    // TODO: better handle this
    self->sim.grid.Imax.x = dim_x - 1;
    self->sim.grid.Imax.y = dim_y - 1;
    self->sim.grid.Imax.z = dim_z - 1;
    self->sim.grid.nIstep.x = dim_x;
    self->sim.grid.nIstep.y = dim_y;
    self->sim.grid.nIstep.z = dim_z;
    self->sim.grid.nIxy  = dim_x * dim_y;
    self->sim.grid.nIxyz = dim_x * dim_y * dim_z;

    Py_RETURN_NONE;
}

////////////////////////////////////////////////////////////////////
//// Getters and setters

//// Setters
// ExecConfig
static int
pypmc_set_n_threads( PyPMC *self, PyObject *value, void *closure )
{
    // TODO: make a macro out of this
    if(! PyLong_Check(value)) {
        PyErr_SetString(PyExc_TypeError,
                        "The n_threads attribute must be an int");
        return -1;
    }

    self->conf.n_threads = PyLong_AsLong(value);

    return 0;
}

static int
pypmc_set_n_iterations( PyPMC *self, PyObject *value, void *closure )
{
    // TODO: make a macro out of this
    if(! PyLong_Check(value)) {
        PyErr_SetString(PyExc_TypeError,
                        "The n_iterations attribute must be an int");
        return -1;
    }

    self->conf.n_iterations = PyLong_AsLong(value);

    return 0;
}

static int
pypmc_set_rand_seed( PyPMC *self, PyObject *value, void *closure )
{
    // TODO: make a macro out of this
    if(! PyLong_Check(value)) {
        PyErr_SetString(PyExc_TypeError,
                        "The rand_seed attribute must be an int");
        return -1;
    }

    self->conf.rand_seed = PyLong_AsLong(value);

    return 0;
}

// Simulation
static int
pypmc_set_n_photons( PyPMC *self, PyObject *value, void *closure )
{
    // TODO: make a macro out of this
    if(! PyLong_Check(value)) {
        PyErr_SetString(PyExc_TypeError,
                        "The n_photons attribute must be an int");
        return -1;
    }

    self->sim.n_photons = PyLong_AsLong(value);

    return 0;
}

static int
pypmc_set_src_pos( PyPMC *self, PyObject *coords, void *closure )
{
    if (! (PyTuple_Check(coords) && PyTuple_Size(coords) == 3))
    {
        PyErr_SetString(PyExc_TypeError,
                        "The attribute must be a tuple with three elements");
        return -1;
    }

    self->sim.src.r.x = (float) PyFloat_AsDouble(PyTuple_GetItem(coords, 0));
    self->sim.src.r.y = (float) PyFloat_AsDouble(PyTuple_GetItem(coords, 1));
    self->sim.src.r.z = (float) PyFloat_AsDouble(PyTuple_GetItem(coords, 2));

    correct_source(&self->sim);

    return 0;
}

static int
pypmc_set_src_dir( PyPMC *self, PyObject *dir_cosines, void *closure )
{
    float3 src_dir;

    if (! (PyTuple_Check(dir_cosines) && PyTuple_Size(dir_cosines) == 3))
    {
        PyErr_SetString(PyExc_TypeError,
                        "The source direction must be a tuple with three elements");
        return -1;
    }

    src_dir.x = (float) PyFloat_AsDouble(PyTuple_GetItem(dir_cosines, 0));
    src_dir.y = (float) PyFloat_AsDouble(PyTuple_GetItem(dir_cosines, 1));
    src_dir.z = (float) PyFloat_AsDouble(PyTuple_GetItem(dir_cosines, 2));

    // Normalize the direction cosine of the source.
    float foo = sqrt(src_dir.x*src_dir.x + src_dir.y*src_dir.y + src_dir.z*src_dir.z);
    src_dir.x /= foo;
    src_dir.y /= foo;
    src_dir.z /= foo;

    self->sim.src.d.x = src_dir.x;
    self->sim.src.d.y = src_dir.y;
    self->sim.src.d.z = src_dir.z;

    return 0;
}

static int
pypmc_set_detectors( PyPMC *self, PyObject *det_list, void *closure )
{
    PyObject *entry, *det_pos, *det_radius;
    Py_ssize_t num_detectors;

    if (! PyList_Check(det_list))
    {
        PyErr_SetString(PyExc_TypeError, "The detectors attribute must be a list");
        return -1;
    }

    // Each entry in the list uniquely identifies a detector.
    num_detectors = PyList_Size(det_list);
    self->sim.det.num = num_detectors;

    // The old detector list must be freed, and a new one built in its place.
    free(self->sim.det.info);
    self->sim.det.info = (int4 *) malloc(num_detectors * sizeof(int4));

    for (int i = 0; i < num_detectors; ++i)
    {
        entry = PyList_GetItem(det_list, i);
        det_pos = PyList_GetItem(entry, 0);
        det_radius = PyList_GetItem(entry, 1);

        self->sim.det.info[i].x = PyLong_AsLong(PyTuple_GetItem(det_pos, 0));
        self->sim.det.info[i].y = PyLong_AsLong(PyTuple_GetItem(det_pos, 1));
        self->sim.det.info[i].z = PyLong_AsLong(PyTuple_GetItem(det_pos, 2));
        self->sim.det.info[i].w = PyLong_AsLong(det_radius);
    }

    return 0;
}

static int
pypmc_set_tissues( PyPMC *self, PyObject *tissue_list, void *closure )
{
    PyObject *entry;
    Py_ssize_t num_tissues;

    if (! PyList_Check(tissue_list))
    {
        PyErr_SetString(PyExc_TypeError, "The tissues attribute must be a list");
        return -1;
    }

    // Each entry in the list uniquely identifies a tissue type.
    num_tissues = PyList_Size(tissue_list);
    self->sim.tiss.num = num_tissues;

    // The old tissue list must be freed, and a new one built in its place.
    free(self->sim.tiss.prop);
    self->sim.tiss.prop = (float4 *) malloc((num_tissues + 1) * sizeof(float4));

    for (int i = 0; i < num_tissues; ++i)
    {
        entry = PyList_GetItem(tissue_list, i);

        self->sim.tiss.prop[i + 1].x = (float) PyFloat_AsDouble(PyTuple_GetItem(entry, 0));
        self->sim.tiss.prop[i + 1].y = (float) PyFloat_AsDouble(PyTuple_GetItem(entry, 1));
        self->sim.tiss.prop[i + 1].z = (float) PyFloat_AsDouble(PyTuple_GetItem(entry, 2));
        self->sim.tiss.prop[i + 1].w = (float) PyFloat_AsDouble(PyTuple_GetItem(entry, 3));
    }

    return 0;
}

static int
pypmc_set_grid_dimensions( PyPMC *self, PyObject *dimensions, void *closure )
{
    PyObject *dim;

    // TODO: verify that every element is a tuple of two elements
    if (! (PyTuple_Check(dimensions) && PyTuple_Size(dimensions) == 3))
    {
        PyErr_SetString(PyExc_TypeError,
                        "The attribute must be a tuple with three elements");
        return -1;
    }

    dim = PyTuple_GetItem(dimensions, 0);
    self->sim.grid.dim.x = PyLong_AsLong(PyTuple_GetItem(dim, 0));
    self->sim.grid.stepr.x = (float) PyFloat_AsDouble(PyTuple_GetItem(dim, 1));

    dim = PyTuple_GetItem(dimensions, 1);
    self->sim.grid.dim.y = PyLong_AsLong(PyTuple_GetItem(dim, 0));
    self->sim.grid.stepr.y = (float) PyFloat_AsDouble(PyTuple_GetItem(dim, 1));

    dim = PyTuple_GetItem(dimensions, 2);
    self->sim.grid.dim.z = PyLong_AsLong(PyTuple_GetItem(dim, 0));
    self->sim.grid.stepr.z = (float) PyFloat_AsDouble(PyTuple_GetItem(dim, 1));

    // Get the minimum dimension.
    self->sim.grid.minstepsize = MIN(self->sim.grid.dim.x,
                                     MIN(self->sim.grid.dim.y,
                                         self->sim.grid.dim.z)); 

    return 0;
}

static int
pypmc_set_fluence_box( PyPMC *self, PyObject *dimensions, void *closure )
{
    PyObject *dim;

    // TODO: verify that every element is a tuple of two elements
    if (! (PyTuple_Check(dimensions) && PyTuple_Size(dimensions) == 3))
    {
        PyErr_SetString(PyExc_TypeError,
                        "The attribute must be a tuple with three elements");
        return -1;
    }

    dim = PyTuple_GetItem(dimensions, 0);
    self->sim.grid.Imin.x = PyLong_AsLong(PyTuple_GetItem(dim, 0));
    self->sim.grid.Imax.x = PyLong_AsLong(PyTuple_GetItem(dim, 1));
    self->sim.grid.nIstep.x = self->sim.grid.Imax.x - self->sim.grid.Imin.x + 1;

    dim = PyTuple_GetItem(dimensions, 1);
    self->sim.grid.Imin.y = PyLong_AsLong(PyTuple_GetItem(dim, 0));
    self->sim.grid.Imax.y = PyLong_AsLong(PyTuple_GetItem(dim, 1));
    self->sim.grid.nIstep.y = self->sim.grid.Imax.y - self->sim.grid.Imin.y + 1;

    dim = PyTuple_GetItem(dimensions, 2);
    self->sim.grid.Imin.z = PyLong_AsLong(PyTuple_GetItem(dim, 0));
    self->sim.grid.Imax.z = PyLong_AsLong(PyTuple_GetItem(dim, 1));
    self->sim.grid.nIstep.z = self->sim.grid.Imax.z - self->sim.grid.Imin.z + 1;

    return 0;
}

//// Getters
// ExecConfig
static PyObject*
pypmc_get_n_threads( PyPMC *self, void *closure )
{
    return PyLong_FromLong(self->conf.n_threads);
}

static PyObject*
pypmc_get_n_iterations( PyPMC *self, void *closure )
{
    return PyLong_FromLong(self->conf.n_iterations);
}

static PyObject*
pypmc_get_rand_seed( PyPMC *self, void *closure )
{
    return PyLong_FromLong(self->conf.rand_seed);
}

// Simulation
static PyObject*
pypmc_get_n_photons( PyPMC *self, void *closure )
{
    return PyLong_FromLong(self->sim.n_photons);
}

static PyObject*
pypmc_get_src_pos( PyPMC *self, void *closure )
{
    PyObject *coords = Py_BuildValue("(fff)", self->sim.src.r.x,
                                              self->sim.src.r.y,
                                              self->sim.src.r.z);

    return coords;
}

static PyObject*
pypmc_get_src_dir( PyPMC *self, void *closure )
{
    PyObject *direction = Py_BuildValue("(fff)", self->sim.src.d.x,
                                                 self->sim.src.d.y,
                                                 self->sim.src.d.z);

    return direction;
}

static PyObject*
pypmc_get_detectors( PyPMC *self, void *closure )
{
    PyObject *det_list = Py_BuildValue("[]");
    PyObject *det_entry, *det_pos, *det_radius;

    for (int i = 0; i < self->sim.det.num; ++i)
    {
        det_pos = Py_BuildValue("(iii)", self->sim.det.info[i].x,
                                         self->sim.det.info[i].y,
                                         self->sim.det.info[i].z);
        det_radius = PyLong_FromLong(self->sim.det.info[i].w);
        det_entry = Py_BuildValue("[NN]", det_pos, det_radius);

        PyList_Append(det_list, det_entry);
    }

    return det_list;
}

static PyObject*
pypmc_get_tissues( PyPMC *self, void *closure )
{
    PyObject *entry;
    PyObject *tissue_list = Py_BuildValue("[]");

    for (int i = 1; i <= self->sim.tiss.num; ++i)
    {
        entry = Py_BuildValue("(ffff)", self->sim.tiss.prop[i].x,
                                        self->sim.tiss.prop[i].y,
                                        self->sim.tiss.prop[i].z,
                                        self->sim.tiss.prop[i].w);

        PyList_Append(tissue_list, entry);
    }

    return tissue_list;
}

static PyObject*
pypmc_get_grid_dimensions( PyPMC *self, void *closure )
{
    PyObject *dim_x, *dim_y, *dim_z;
    PyObject *dimensions;

    dim_x = Py_BuildValue("(fi)", self->sim.grid.stepr.x, self->sim.grid.dim.x);
    dim_y = Py_BuildValue("(fi)", self->sim.grid.stepr.y, self->sim.grid.dim.y);
    dim_z = Py_BuildValue("(fi)", self->sim.grid.stepr.z, self->sim.grid.dim.z);

    dimensions = Py_BuildValue("(NNN)", dim_x, dim_y, dim_z);

    return dimensions;
}

static PyObject*
pypmc_get_fluence_box( PyPMC *self, void *closure )
{
    PyObject *dim_x, *dim_y, *dim_z;
    PyObject *dimensions;

    dim_x = Py_BuildValue("(ii)", self->sim.grid.Imin.x, self->sim.grid.Imax.x);
    dim_y = Py_BuildValue("(ii)", self->sim.grid.Imin.y, self->sim.grid.Imax.y);
    dim_z = Py_BuildValue("(ii)", self->sim.grid.Imin.z, self->sim.grid.Imax.z);

    dimensions = Py_BuildValue("(NNN)", dim_x, dim_y, dim_z);

    return dimensions;
}

static PyObject*
pypmc_get_tissueArray( Simulation sim, float *tissueArray )
{
    uint photonIndex, k;
    int detIndex, tissueIndex;
    PyObject *py_tissueArray = Py_BuildValue("[]");

    if( sim.det.num != 0 )
    {
        for( photonIndex = 0; photonIndex < sim.n_photons; photonIndex++ )
        {
            for( detIndex = 0; detIndex < sim.det.num; detIndex++ )
            {
                if( bitset_get(sim.detHit, photonIndex, detIndex) == 1 )
                {
                    for( tissueIndex = 1; tissueIndex <= sim.tiss.num; tissueIndex++ )
                    {
                        k = MAD_HASH((photonIndex << 5) | tissueIndex);

                        PyList_Append(py_tissueArray, PyFloat_FromDouble(tissueArray[k]));
                    }
                }
            }
        }
    }

    return py_tissueArray;
}

//// end of getters and setters
////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////
//// Miscellaneous/Extension-related
static PyGetSetDef pypmc_getsetters[] = {
    // Execution parameters (ExecConfig) 
    {"n_threads",
     (getter) pypmc_get_n_threads, (setter) pypmc_set_n_threads, 
     "number of CUDA threads used in the simulation", NULL},
    {"n_iterations",
     (getter) pypmc_get_n_iterations, (setter) pypmc_set_n_iterations, 
     "number of CUDA iterations used in the simulation", NULL},
    {"rand_seed",
     (getter) pypmc_get_rand_seed, (setter) pypmc_set_rand_seed, 
     "number of CUDA iterations used in the simulation", NULL},

    // Simulation parameters (Simulation)
    {"n_photons",
     (getter) pypmc_get_n_photons, (setter) pypmc_set_n_photons, 
     "number of photons simulated", NULL},
    {"src_pos",
     (getter) pypmc_get_src_pos, (setter) pypmc_set_src_pos, 
     "euclidean position of the source (automatically corrected to be at an interface)", NULL},
    {"src_dir",
     (getter) pypmc_get_src_dir, (setter) pypmc_set_src_dir, 
     "direction cosines of the source", NULL},
    {"detectors",
     (getter) pypmc_get_detectors, (setter) pypmc_set_detectors, 
     "list of detectors (their position and radius)", NULL},
    {"tissues",
     (getter) pypmc_get_tissues, (setter) pypmc_set_tissues, 
     "list of tissues (their optical properties)", NULL},
    {"grid_dimensions",
     (getter) pypmc_get_grid_dimensions, (setter) pypmc_set_grid_dimensions, 
     "the grid's dimensions (voxel size and number of voxels in each direction)", NULL},
    {"fluence_box",
     (getter) pypmc_get_fluence_box, (setter) pypmc_set_fluence_box, 
     "the fluence box's vertices (in each direction)", NULL},
/*
    {"min_time",
     (getter) pypmc_get_min_time, (setter) pypmc_set_min_time, 
     "minimum duration of simulation for a given photon to be accounted in the fluence calculation", NULL},
    {"max_time",
     (getter) pypmc_get_max_time, (setter) pypmc_set_max_time, 
     "maximum duration of simulation for a given photon to be accounted in the fluence calculation", NULL},
    {"time_step",
     (getter) pypmc_get_time_step, (setter) pypmc_set_time_step, 
     "simulation time step", NULL},
*/

    {NULL} /* Sentinel */
};

static PyMemberDef pypmc_members[] = {
    // Simulation results
    {"pathlength", T_OBJECT_EX, offsetof(PyPMC, py_pathlength), READONLY,
     "the distance travelled by each photon in each type of tissue"},
    {"momentum_transfer", T_OBJECT_EX, offsetof(PyPMC, py_momentum_transfer), READONLY,
     "momentum transfer"},

    {NULL} /* Sentinel */
};

static PyMethodDef pypmc_methods[] = {
    {"run_simulation", (PyCFunction) pypmc_run, METH_NOARGS,
     "Does what it says on the tin."},
    {"pull_results", (PyCFunction) pypmc_pull_results, METH_NOARGS,
     "Transfers the simulation results to the host memory."},
    {"push_parameters", (PyCFunction) pypmc_push_parameters, METH_NOARGS,
     "Transfers the simulation parameters to the gpu memory."},
    {"write_to_disk", (PyCFunction) pypmc_write_to_disk, METH_VARARGS,
     "Saves the simulation results to disk, the old-fashioned way."},
    {"load_medium", (PyCFunction) pypmc_load_medium, METH_VARARGS,
     "Load tridimensional medium"},

    {NULL}  /* Sentinel */
};

static PyTypeObject pypmc_Type = {
    PyVarObject_HEAD_INIT(NULL, 0)
    "pypmc.pypmc",                              /* tp_name */
    sizeof(PyPMC),                              /* tp_basicsize */
    0,                                          /* tp_itemsize */
    (destructor) pypmc_dealloc,                 /* tp_dealloc */
    0,                                          /* tp_print */
    0,                                          /* tp_getattr */
    0,                                          /* tp_setattr */
    0,                                          /* tp_compare */
    0,                                          /* tp_repr */
    0,                                          /* tp_as_number */
    0,                                          /* tp_as_sequence */
    0,                                          /* tp_as_mapping */
    0,                                          /* tp_hash */
    0,                                          /* tp_call */
    0,                                          /* tp_str */
    0,                                          /* tp_getattro */
    0,                                          /* tp_setattro */
    0,                                          /* tp_as_buffer */
    Py_TPFLAGS_DEFAULT | Py_TPFLAGS_BASETYPE,   /* tp_flags */
    "pypmc objects",                            /* tp_doc */
    0,                                          /* tp_traverse */
    0,                                          /* tp_clear */
    0,                                          /* tp_richcompare */
    0,                                          /* tp_weaklistoffset */
    0,                                          /* tp_iter */
    0,                                          /* tp_iternext */
    pypmc_methods,                              /* tp_methods */
    pypmc_members,                              /* tp_members */
    pypmc_getsetters,                           /* tp_getset */
    0,                                          /* tp_base */
    0,                                          /* tp_dict */
    0,                                          /* tp_descr_get */
    0,                                          /* tp_descr_set */
    0,                                          /* tp_dictoffset */
    (initproc) pypmc_init,                      /* tp_init */
    0,                                          /* tp_alloc */
    pypmc_new,                                  /* tp_new */
};

static PyModuleDef pypmc_module = {
    PyModuleDef_HEAD_INIT,
    "pypmc",
    "Python bindings for PMC.",
    -1,
    NULL, NULL, NULL, NULL, NULL
};

PyMODINIT_FUNC
PyInit_pypmc(void)
{
    PyObject *m;

    pypmc_Type.tp_new = PyType_GenericNew; 
    if (PyType_Ready(&pypmc_Type) < 0)
        return NULL;

    m = PyModule_Create(&pypmc_module);

    // Ensure that the module was correctly initialized.
    if (m == NULL)
        return NULL;

    // Protect the python type from being prematurely garbage collected.
    Py_INCREF(&pypmc_Type);

    // Load module into the interpreter.
    PyModule_AddObject(m, "PyPMC", (PyObject *) &pypmc_Type);

    return m;
}
